
#include<hip/hip_runtime.h>
#include<algorithm>
#include<iostream>
#include<assert.h>
#include<stdlib.h>
#include<stdio.h>
#include<time.h>
#include<math.h>
#include<vector>

// CUDA kernel for vector addition

__global__ void vecAdd(float *out, float *a, float *b, int n) {
    
    // Calculate global thread ID 
    
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Vector boundary gaurd
    
    if (i < n){

        // Each thread adds a single element
        
        out[i] = a[i] + b[i];
        }
}

// Check results to make sure they are correct
void verify_results(std :: vector<float> &out, std :: vector<float> &a, std :: vector<float> &b, int n) {
    for (int i = 0; i < n; i++) {
        assert(out[i] == a[i] + b[i]);
    }
}

int main() {
    
    // Set up device
    
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);
    
    // Set up data size of vectors
    
    int n = 1 << 20;
    
    // Allocate host memory
    
    size_t size_a = n * sizeof(float);
    size_t size_b = n * sizeof(float);
    size_t size_out = n * sizeof(float);
    
    // Allocate device memory
    
    float *d_a, *d_b, *d_out;
    
    hipMalloc((void **) &d_a, size_a);
    hipMalloc((void **) &d_b, size_b);
    hipMalloc((void **) &d_out, size_out);
    
    // Initialize host memory
    
    float *h_a = (float *)malloc(size_a);
    float *h_b = (float *)malloc(size_b);
    float *h_out = (float *)malloc(size_out);
    
    // Initialize host vectors
    
    for (int i = 0; i < n; i++) {
        h_a[i] = 1.0f;
        h_b[i] = 2.0f;
    }
    
    // Initialize device vectors
    
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    
    // Launch vector addition kernel
    vecAdd<<<1, n>>>(d_out, d_a, d_b, n);
    
    // Copy result from device to host
    
    hipMemcpy(h_out, d_out, size_out, hipMemcpyDeviceToHost);
    
    // Verify results
    
    std :: vector<float> out(n);
    std :: vector<float> a(n);
    std :: vector<float> b(n);
    
    hipMemcpy(out.data(), d_out, size_out, hipMemcpyDeviceToHost);
    hipMemcpy(a.data(), d_a, size_a, hipMemcpyDeviceToHost);
    hipMemcpy(b.data(), d_b, size_b, hipMemcpyDeviceToHost);
    
    verify_results(out, a, b, n);
    
    // Free device memory
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    
    // Free host memory
    
    free(h_a);
    free(h_b);
    free(h_out);
    
    return 0;

}